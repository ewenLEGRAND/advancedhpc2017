#include "hip/hip_runtime.h"

#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>
#include <math.h>

#define ACTIVE_THREADS 4


int main(int argc, char **argv) {
    printf("USTH ICT Master 2017, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum) {
        case 1:
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
	    labwork.labwork1_OpenMP();
            labwork.saveOutputImage("labwork2-openmp-out.jpg");
            printf("labwork 1 OpenMP ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec()); 
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
            break;
        case 4:
            labwork.labwork4_GPU();
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
            break;
        case 5:
            labwork.labwork5_CPU();
            labwork.saveOutputImage("labwork5-cpu-out.jpg");
            labwork.labwork5_GPU();
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
    printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {		// let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }    }
}

void Labwork::labwork1_OpenMP() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    omp_set_num_threads(12); // blocs size
    #pragma omp parallel for schedule(dynamic, 3)
    for (int j = 0; j < 100; j++) {             // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {

    int devCount;
    hipGetDeviceCount(&devCount);
    printf("Device number : %d\n",devCount);
    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        printf("Major : %d\n",props.major);
        printf("Total global memory : %zu\n",props.totalGlobalMem);
        printf("Shared memory peu block : %zu\n",props.sharedMemPerBlock);
        //printf("%s\n",props.totalConstMem);
        printf("Registers per block : %d\n",props.regsPerBlock);
	printf("Clock rate : %d\n",props.clockRate);
	printf("Multiprocessor count : %d\n",props.multiProcessorCount);
	printf("Memory Bus Width : %d\n",props.memoryBusWidth);
	printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*props.memoryClockRate*(props.memoryBusWidth/8)/1.0e6);// source :devblogs.nvidia
        printf("Warp size : %d\n",props.warpSize);
        printf("Max threads per blocks : %d\n",props.maxThreadsPerBlock);
        printf("Max threads dimension :\n 1 : %7d\n2 : %7d\n3 : %7d\n",props.maxThreadsDim[0],props.maxThreadsDim[1],props.maxThreadsDim[2]);
        printf("Max grid size :\n 1 : %7d\n2 : %7d\n3 : %7d\n",props.maxGridSize[0],props.maxGridSize[1],props.maxGridSize[2]);
        printf("\n\n\n");
    }   
}


__global__ void imageComputeLab3(uchar3 *devImage, uchar3 *devOutputImage){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
            devOutputImage[tid].x = (char) (((int) devImage[tid].x + (int) devImage[tid].y +
                                          (int) devImage[tid].z) / 3);
            devOutputImage[tid].y = devOutputImage[tid].x;
            devOutputImage[tid].z = devOutputImage[tid].x;
}

void Labwork::labwork3_GPU() {
            uchar3 *devImage;
            uchar3 *devOutputImage;
            uchar3 *hostOutputImage;
            int pixelCount =inputImage->width *inputImage->height;
            int blockSize = 1024;
            int numBlock = pixelCount / blockSize;

            hipMalloc(&devImage, pixelCount * 3);
            hipMalloc(&devOutputImage, pixelCount * 3);
            hostOutputImage = (uchar3 *) malloc(pixelCount * 3);

            hipMemcpy(devImage, inputImage->buffer,pixelCount * sizeof(uchar3),hipMemcpyHostToDevice); // Memory transfert

            imageComputeLab3<<<numBlock, blockSize>>>(devImage,devOutputImage); // Kernel

            hipMemcpy(hostOutputImage, devOutputImage,pixelCount * sizeof(uchar3),hipMemcpyDeviceToHost);
	    outputImage = (char *)hostOutputImage;
            hipFree(devImage);   
	    hipFree(devOutputImage);
}

__global__ void imageComputeLab4(uchar3 *devImage, uchar3 *devOutputImage,int width){
            int x = threadIdx.x + blockIdx.x * blockDim.x;
	    int y = threadIdx.y + blockIdx.y * blockDim.y;
	    int tid = y * width +x;
            devOutputImage[tid].x = (char) ((int) (ceil((float) devImage[tid].x) + (int) ceil((float) devImage[tid].y) +
                                         (int) ceil((float) devImage[tid].z)) / 3);
            devOutputImage[tid].y = devOutputImage[tid].x;
            devOutputImage[tid].z = devOutputImage[tid].x;
}

void Labwork::labwork4_GPU() {
            uchar3 *devImage;
            uchar3 *devOutputImage;
            uchar3 *hostOutputImage;
            dim3 blockSize = dim3(32,32);
            int pixelCount =inputImage->width *inputImage->height;	   
	    int width = inputImage->width;
	    dim3 gridSize = dim3(inputImage->width/blockSize.x,inputImage->height/blockSize.y);

            hipMalloc(&devImage, pixelCount * 3);
            hipMalloc(&devOutputImage, pixelCount * 3);
            hostOutputImage = (uchar3 *) malloc(pixelCount * 3);

            hipMemcpy(devImage, inputImage->buffer,pixelCount * sizeof(uchar3),hipMemcpyHostToDevice); // Memory transfert

            imageComputeLab4<<<gridSize, blockSize>>>(devImage,devOutputImage,width); // Kernel
	
            hipMemcpy(hostOutputImage, devOutputImage,pixelCount * sizeof(uchar3),hipMemcpyDeviceToHost);
            outputImage = (char *)hostOutputImage;
            hipFree(devImage);   
	    hipFree(devOutputImage);
}



// CPU implementation of Gaussian Blur
void Labwork::labwork5_CPU() {
    int kernel[] = { 0, 0, 1, 2, 1, 0, 0,
                     0, 3, 13, 22, 13, 3, 0,
                     1, 13, 59, 97, 59, 13, 1,
                     2, 22, 97, 159, 97, 22, 2,
                     1, 13, 59, 97, 59, 13, 1,
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0 };
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = (char*) malloc(pixelCount * sizeof(char) * 3);
    for (int row = 0; row < inputImage->height; row++) {
        for (int col = 0; col < inputImage->width; col++) {
            int sum = 0;
            int c = 0;
            for (int y = -3; y <= 3; y++) {
                for (int x = -3; x <= 3; x++) {
                    int i = col + x;
                    int j = row + y;
                    if (i < 0) continue;
                    if (i >= inputImage->width) continue;
                    if (j < 0) continue;
                    if (j >= inputImage->height) continue;
                    int tid = j * inputImage->width + i;
                    unsigned char gray = (inputImage->buffer[tid * 3] + inputImage->buffer[tid * 3 + 1] + inputImage->buffer[tid * 3 + 2])/3;
                    int coefficient = kernel[(y+3) * 7 + x + 3];
                    sum = sum + gray * coefficient;
                    c += coefficient;
                }
            }
            sum /= c;
            int posOut = row * inputImage->width + col;
            outputImage[posOut * 3] = outputImage[posOut * 3 + 1] = outputImage[posOut * 3 + 2] = sum;
        }
    }
}


__global__ void filterLab5(uchar3 *filterDevOutputImage, uchar3 *devOutputImage,int width, int height){
            int x = threadIdx.x + blockIdx.x * blockDim.x;
            int y = threadIdx.y + blockIdx.y * blockDim.y;
            int tid = y * width +x;
	    int filter[] = { 0, 0, 1, 2, 1, 0, 0,
                     0, 3, 13, 22, 13, 3, 0,
                     1, 13, 59, 97, 59, 13, 1,
                     2, 22, 97, 159, 97, 22, 2,
                     1, 13, 59, 97, 59, 13, 1,
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0 };


	    float outputPixel = 0;
	    for (int i = -3; i <= 3; i++)
	    {
		for (int j = -3; j <= 3; j++)
		{
		    if ( (x + i) < 0)	// left side
			continue;
		    if ( (x + i) >= width )	// right side
		        continue;
		    if ((y + j) < 0)	// top side
			continue;
		    if ((y + j) >= height )	// bottom side
			continue;
		    int localtid = (x+i)+ (y+j)*width;
		    unsigned char grey = (devOutputImage[localtid].x + devOutputImage[localtid].y + devOutputImage[localtid].z)/3;
                    int coefficient = filter[(j+3) * 7 + i + 3];
		    outputPixel += coefficient * grey;
		   
	        }
	    }
            filterDevOutputImage[tid].x = outputPixel/1003;
	    filterDevOutputImage[tid].y = filterDevOutputImage[tid].z = filterDevOutputImage[tid].x;
}

__global__ void greyScalingLab5(uchar3 *devImage, uchar3 *devOutputImage, int width){
            int x = threadIdx.x + blockIdx.x * blockDim.x;
            int y = threadIdx.y + blockIdx.y * blockDim.y;
            int tid = y * width + x;

            devOutputImage[tid].x = (char) ((int) (ceil((float) devImage[tid].x) + (int) ceil((float) devImage[tid].y) +
                                         (int) ceil((float) devImage[tid].z)) / 3);
            devOutputImage[tid].y = devOutputImage[tid].x;
            devOutputImage[tid].z = devOutputImage[tid].x;

}


void Labwork::labwork5_GPU() {
/*
	    float GaussianFilter[7][7] ={
		{1,4,7,10,7,4,1},
		{4,12,26,33,26,12,4},
		{7,26,55,71,55,26,7},
		{10,33,71,91,71,33,10},
	        {7,26,55,71,55,26,7},
		{4,12,26,33,26,12,4},
		{1,4,7,10,7,4,1},
	    }; // Sum equal to 1115
*/
/*	    int filter[] = { 0, 0, 1, 2, 1, 0, 0,
                     0, 3, 13, 22, 13, 3, 0,
                     1, 13, 59, 97, 59, 13, 1,
                     2, 22, 97, 159, 97, 22, 2,
                     1, 13, 59, 97, 59, 13, 1,
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0 };
 */     
	    uchar3 *devImage;
	    uchar3 *devOutputImage;
            uchar3 *hostOutputImageFilter;
	    uchar3 *filterDevOutputImage;

            dim3 blockSize = dim3(256,256);
            int pixelCount =inputImage->width *inputImage->height;
            int width = inputImage->width;
	    int height = inputImage->height;
            dim3 gridSize = dim3(inputImage->width/blockSize.x,inputImage->height/blockSize.y);

	    hostOutputImageFilter = (uchar3 *) malloc(pixelCount*3);
            hipMalloc(&devImage, pixelCount * 3);
            hipMalloc(&devOutputImage, pixelCount*3);      
	    hipMalloc(&filterDevOutputImage, pixelCount*3);

            hipMemcpy(devImage, inputImage->buffer,pixelCount * sizeof(uchar3),hipMemcpyHostToDevice); // Memory transfert

            greyScalingLab5<<<gridSize, blockSize>>>(devImage,devOutputImage,width); // Kernel greyscaling

            filterLab5<<<gridSize, blockSize>>>(filterDevOutputImage,devOutputImage,width, height); // Kernel
	    
            hipMemcpy(hostOutputImageFilter, filterDevOutputImage, pixelCount*3, hipMemcpyDeviceToHost);

	    outputImage = (char *)hostOutputImageFilter;
            hipFree(devImage);
            hipFree(devOutputImage);
            hipFree(filterDevOutputImage);

}




__global__ void binarisationLab6(uchar3 *binarisationDevOutputImage, uchar3 *devOutputImage,int width){
            int x = threadIdx.x + blockIdx.x * blockDim.x;
            int y = threadIdx.y + blockIdx.y * blockDim.y;
            int tid = y * width + x;
	    int therehold = 256/2;
	    unsigned char grey = devOutputImage[tid].x;

	    if(grey<therehold){
		binarisationDevOutputImage[tid].x = 0;
	    }
	    else{
		binarisationDevOutputImage[tid].x = 255;
	    }

	    binarisationDevOutputImage[tid].y = binarisationDevOutputImage[tid].z = binarisationDevOutputImage[tid].x;
	    
}

__global__ void brightnessLab6(uchar3 *binarisationDevOutputImage, uchar3 *devOutputImage,int width){
            int x = threadIdx.x + blockIdx.x * blockDim.x;
            int y = threadIdx.y + blockIdx.y * blockDim.y;
            int tid = y * width + x;
            unsigned char grey = devOutputImage[tid].x;

            binarisationDevOutputImage[tid].x = grey + (int) 0.2*grey;
            binarisationDevOutputImage[tid].y = binarisationDevOutputImage[tid].z = binarisationDevOutputImage[tid].x;

}



__global__ void greyScalingLab6(uchar3 *devImage, uchar3 *devOutputImage, int width){
            int x = threadIdx.x + blockIdx.x * blockDim.x;
            int y = threadIdx.y + blockIdx.y * blockDim.y;
            int tid = y * width + x;

            devOutputImage[tid].x = (char) ((int) (ceil((float) devImage[tid].x) + (int) ceil((float) devImage[tid].y) +
                                         (int) ceil((float) devImage[tid].z)) / 3);
            devOutputImage[tid].y = devOutputImage[tid].x;
            devOutputImage[tid].z = devOutputImage[tid].x;
}


void Labwork::labwork6_GPU() {
            uchar3 *devImage;
            uchar3 *devOutputImage;
            uchar3 *hostOutputImageFilter;
            uchar3 *binarisationDevOutputImage;

            dim3 blockSize = dim3(32,32);
            int pixelCount =inputImage->width *inputImage->height;
            int width = inputImage->width;
            dim3 gridSize = dim3(inputImage->width/blockSize.x,inputImage->height/blockSize.y);

            hostOutputImageFilter = (uchar3 *) malloc(pixelCount*3);
            hipMalloc(&devImage, pixelCount * 3);
            hipMalloc(&devOutputImage, pixelCount*3);
            hipMalloc(&binarisationDevOutputImage, pixelCount*3);


            hipMemcpy(devImage, inputImage->buffer,pixelCount * sizeof(uchar3),hipMemcpyHostToDevice); // Memory transfert

            greyScalingLab6<<<gridSize, blockSize>>>(devImage,devOutputImage,width); // Kernel greyscaling

            binarisationLab6<<<gridSize, blockSize>>>(binarisationDevOutputImage,devOutputImage,width); // Kernel
	    
	    //brightnessLab6<<<gridSize, blockSize>>>(binarisationDevOutputImage,devOutputImage,width); // Kernel
            
	    hipMemcpy(hostOutputImageFilter, binarisationDevOutputImage, pixelCount*3, hipMemcpyDeviceToHost);

            outputImage = (char *)hostOutputImageFilter;
            hipFree(devImage);
            hipFree(devOutputImage);
            hipFree(binarisationDevOutputImage);

}






void Labwork::labwork7_GPU() {

}

void Labwork::labwork8_GPU() {

}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU() {

}
