#include "hip/hip_runtime.h"
#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>
#include <math.h>

#define ACTIVE_THREADS 4


int main(int argc, char **argv) {
    printf("USTH ICT Master 2017, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum) {
        case 1:
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
	    labwork.labwork1_OpenMP();
            labwork.saveOutputImage("labwork2-openmp-out.jpg");
            printf("labwork 1 OpenMP ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec()); 
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
            break;
        case 4:
            labwork.labwork4_GPU();
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
            break;
        case 5:
            labwork.labwork5_CPU();
            labwork.saveOutputImage("labwork5-cpu-out.jpg");
            labwork.labwork5_GPU();
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
    printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {		// let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }    }
}

void Labwork::labwork1_OpenMP() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    omp_set_num_threads(12); // blocs size
    #pragma omp parallel for schedule(dynamic, 3)
    for (int j = 0; j < 100; j++) {             // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {

    int devCount;
    hipGetDeviceCount(&devCount);
    printf("Device number : %d\n",devCount);
    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        printf("Major : %d\n",props.major);
        printf("Total global memory : %zu\n",props.totalGlobalMem);
        printf("Shared memory peu block : %zu\n",props.sharedMemPerBlock);
        //printf("%s\n",props.totalConstMem);
        printf("Registers per block : %d\n",props.regsPerBlock);
	printf("Clock rate : %d\n",props.clockRate);
	printf("Multiprocessor count : %d\n",props.multiProcessorCount);
	printf("Memory Bus Width : %d\n",props.memoryBusWidth);
	printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*props.memoryClockRate*(props.memoryBusWidth/8)/1.0e6);// source :devblogs.nvidia
        printf("Warp size : %d\n",props.warpSize);
        printf("Max threads per blocks : %d\n",props.maxThreadsPerBlock);
        printf("Max threads dimension :\n 1 : %7d\n2 : %7d\n3 : %7d\n",props.maxThreadsDim[0],props.maxThreadsDim[1],props.maxThreadsDim[2]);
        printf("Max grid size :\n 1 : %7d\n2 : %7d\n3 : %7d\n",props.maxGridSize[0],props.maxGridSize[1],props.maxGridSize[2]);
        printf("\n\n\n");
    }   
}


__global__ void imageComputeLab3(uchar3 *devImage, uchar3 *devOutputImage){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
            devOutputImage[tid].x = (char) (((int) devImage[tid].x + (int) devImage[tid].y +
                                          (int) devImage[tid].z) / 3);
            devOutputImage[tid].y = devOutputImage[tid].x;
            devOutputImage[tid].z = devOutputImage[tid].x;
}

void Labwork::labwork3_GPU() {
            uchar3 *devImage;
            uchar3 *devOutputImage;
            uchar3 *hostOutputImage;
            int pixelCount =inputImage->width *inputImage->height;
            int blockSize = 1024;
            int numBlock = pixelCount / blockSize;

            hipMalloc(&devImage, pixelCount * 3);
            hipMalloc(&devOutputImage, pixelCount * 3);
            hostOutputImage = (uchar3 *) malloc(pixelCount * 3);

            hipMemcpy(devImage, inputImage->buffer,pixelCount * sizeof(uchar3),hipMemcpyHostToDevice); // Memory transfert

            imageComputeLab3<<<numBlock, blockSize>>>(devImage,devOutputImage); // Kernel

            hipMemcpy(hostOutputImage, devOutputImage,pixelCount * sizeof(uchar3),hipMemcpyDeviceToHost);
	    outputImage = (char *)hostOutputImage;
            hipFree(devImage);   
	    hipFree(devOutputImage);
}

__global__ void imageComputeLab4(uchar3 *devImage, uchar3 *devOutputImage,int width){
            int x = threadIdx.x + blockIdx.x * blockDim.x;
	    int y = threadIdx.y + blockIdx.y * blockDim.y;
	    int tid = y * width +x;
            devOutputImage[tid].x = (char) ((int) (ceil((float) devImage[tid].x) + (int) ceil((float) devImage[tid].y) +
                                         (int) ceil((float) devImage[tid].z)) / 3);
            devOutputImage[tid].y = devOutputImage[tid].x;
            devOutputImage[tid].z = devOutputImage[tid].x;
}

void Labwork::labwork4_GPU() {
            uchar3 *devImage;
            uchar3 *devOutputImage;
            uchar3 *hostOutputImage;
            dim3 blockSize = dim3(32,32);
            int pixelCount =inputImage->width *inputImage->height;	   
	    int width = inputImage->width;
	    dim3 gridSize = dim3(inputImage->width/blockSize.x,inputImage->height/blockSize.y);

            hipMalloc(&devImage, pixelCount * 3);
            hipMalloc(&devOutputImage, pixelCount * 3);
            hostOutputImage = (uchar3 *) malloc(pixelCount * 3);

            hipMemcpy(devImage, inputImage->buffer,pixelCount * sizeof(uchar3),hipMemcpyHostToDevice); // Memory transfert

            imageComputeLab4<<<gridSize, blockSize>>>(devImage,devOutputImage,width); // Kernel

            hipMemcpy(hostOutputImage, devOutputImage,pixelCount * sizeof(uchar3),hipMemcpyDeviceToHost);
            outputImage = (char *)hostOutputImage;
            hipFree(devImage);   
	    hipFree(devOutputImage);
}

__global__ void greyScalingLab5(uchar3 *devImage, uchar3 *devOutputImage){
            int x = threadIdx.x + blockIdx.x * blockDim.x;
            int y = threadIdx.y + blockIdx.y * blockDim.y;
           int tid = y * width +x;
            devOutputImage[tid].x = (char) ((int) (ceil((float) devImage[tid].x) + (int) ceil((float) devImage[tid].y) +
                                         (int) ceil((float) devImage[tid].z)) / 3);
            devOutputImage[tid].y = devOutputImage[tid].x;
            devOutputImage[tid].z = devOutputImage[tid].x;
}

__global__ void filterLab5(uchar3 *devImage, uchar3 *devOutputImage,int * filter,int width){
            int x = threadIdx.x + blockIdx.x * blockDim.x;
            int y = threadIdx.y + blockIdx.y * blockDim.y;
            int tid = y * width +x;

	    float sum = 0;
	    float value = 0;
	    for (int i = -3; i <= 3; i++)	// row wise
		for (int j = -3; j <= 3; j++)	// col wise
		{
		    if (blockIdx.x == 0 && (threadIdx.x + i) < 0)	// left apron
			value = 0;
		    else if ( blockIdx.x == (gridDim.x - 1) &&	(threadIdx.x + i) > blockDim.x-1 )	// right apron
		        value = 0;
		    else 
		    { 
			if (blockIdx.y == 0 && (threadIdx.y + j) < 0)	// top apron
				value = 0;
			else if ( blockIdx.y == (gridDim.y - 1) && (threadIdx.y + j) > blockDim.y-1 )	// bottom apron
				value = 0;
			else	// safe case
				value = devImage[tid + i + j * width];
	            } 
		    sum += value * filter[3 + i] * filter[3 + j];
	        }
	        devOutputImage[tid] = sum;
	    }
}


// CPU implementation of Gaussian Blur
void Labwork::labwork5_CPU() {
    int kernel[] = { 0, 0, 1, 2, 1, 0, 0,  
                     0, 3, 13, 22, 13, 3, 0,  
                     1, 13, 59, 97, 59, 13, 1,  
                     2, 22, 97, 159, 97, 22, 2,  
                     1, 13, 59, 97, 59, 13, 1,  
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0 };
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = (char*) malloc(pixelCount * sizeof(char) * 3);
    for (int row = 0; row < inputImage->height; row++) {
        for (int col = 0; col < inputImage->width; col++) {
            int sum = 0;
            int c = 0;
            for (int y = -3; y <= 3; y++) {
                for (int x = -3; x <= 3; x++) {
                    int i = col + x;
                    int j = row + y;
                    if (i < 0) continue;
                    if (i >= inputImage->width) continue;
                    if (j < 0) continue;
                    if (j >= inputImage->height) continue;
                    int tid = j * inputImage->width + i;
                    unsigned char gray = (inputImage->buffer[tid * 3] + inputImage->buffer[tid * 3 + 1] + inputImage->buffer[t$
                    int coefficient = kernel[(y+3) * 7 + x + 3];
                    sum = sum + gray * coefficient;
                    c += coefficient;
                }
            }
            sum /= c;
            int posOut = row * inputImage->width + col;
            outputImage[posOut * 3] = outputImage[posOut * 3 + 1] = outputImage[posOut * 3 + 2] = sum;
        }
    }
}



void Labwork::labwork5_GPU() {
    float GaussianFilter[7][7] ={
	{1,4,7,10,7,4,1},
	{4,12,26,33,26,12,4},
	{7,26,55,71,55,26,7},
	{10,33,71,91,71,33,10},
        {7,26,55,71,55,26,7},
        {4,12,26,33,26,12,4},
	{1,4,7,10,7,4,1},
    }; // Sum equal to 1115
}

void Labwork::labwork6_GPU() {

}

void Labwork::labwork7_GPU() {

}

void Labwork::labwork8_GPU() {

}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU() {

}
